#include <unittest/unittest.h>
#include <thrust/execution_policy.h>
#include <thrust/system/cuda/detail/util.h>

#include <thread>

void verify_stream()
{
  auto exec = thrust::device;
  auto stream = thrust::cuda_cub::stream(exec);
#ifdef CUDA_API_PER_THREAD_DEFAULT_STREAM
  ASSERT_EQUAL(stream, hipStreamPerThread);
#else
  ASSERT_EQUAL(stream, (hipStream_t)hipStreamDefault);
#endif
}

void TestPerThreadDefaultStream()
{
  verify_stream();

  std::thread t(verify_stream);
  t.join();
}
DECLARE_UNITTEST(TestPerThreadDefaultStream);
